#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

 
 __global__ void __print_kernel__ ()
 {
    printf("GPU says, Hello world! \n");
 }
 
 extern "C" void print_gpu()
{
     __print_kernel__ <<<1,1>>> ();
     hipDeviceSynchronize();
}
